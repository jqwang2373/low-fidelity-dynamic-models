#include "hip/hip_runtime.h"
#include <algorithm>
#include <cassert>
#include <cmath>
#include <iostream>
#include <stdint.h>
#include <vector>

#include "dof18_gpu.cuh"
#include "dof18_halfImplicit_gpu.cuh"

using namespace d18;
// ======================================================================================================================
d18SolverHalfImplicitGPU::d18SolverHalfImplicitGPU(unsigned int total_num_vehicles)
    : m_step(0.001), m_output(false), m_csv(" "), m_vehicle_count_tracker_params(0), m_vehicle_count_tracker_states(0) {
    m_total_num_vehicles = total_num_vehicles;

    // Allocate memory for the simData and simStates
    CHECK_CUDA_ERROR(hipMallocManaged((void**)&m_sim_data, sizeof(d18::SimData) * m_total_num_vehicles));
    CHECK_CUDA_ERROR(hipMallocManaged((void**)&m_sim_data_nr, sizeof(d18::SimDataNr) * m_total_num_vehicles));
    CHECK_CUDA_ERROR(hipMallocManaged((void**)&m_sim_states, sizeof(d18::SimState) * m_total_num_vehicles));
    CHECK_CUDA_ERROR(hipMallocManaged((void**)&m_sim_states_nr, sizeof(d18::SimStateNr) * m_total_num_vehicles));
}
d18SolverHalfImplicitGPU::~d18SolverHalfImplicitGPU() {
    // Only need to delete the memory of the simData and simStates of the respective tire as the rest of the memory is
    // freed as soon as we have information of what tire the user is using
    if (m_tire_type == TireType::TMeasy) {
        hipFree(m_sim_data);
        hipFree(m_sim_states);
        hipFree(device_response);
        delete[] host_response;
    } else {
        hipFree(m_sim_data_nr);
        hipFree(m_sim_states_nr);
        hipFree(device_response_nr);
        delete[] host_response_nr;
    }
}
// ======================================================================================================================
// Construct the solver using path to vehicle parameters, tire parameters, number of vehicles and driver
__host__ void d18SolverHalfImplicitGPU::Construct(const std::string& vehicle_params_file,
                                                  const std::string& tire_params_file,
                                                  unsigned int num_vehicles,
                                                  const std::string& driver_inputs_file) {
    // Check if num_vehicles added is less than the total number of vehicles
    assert((num_vehicles + m_vehicle_count_tracker_params <= m_total_num_vehicles) &&
           "Number of vehicles added makes the vehicle count greater than the total number of vehicles");
    // If there is no tire type specified, then use TMeasy
    m_tire_type = TireType::TMeasy;
    // Because of this, we free the memory of the TMeasyNR tire
    hipFree(m_sim_data_nr);
    hipFree(m_sim_states_nr);
    hipFree(device_response_nr);
    // Set these to nullptr so that we don't try to free them again in the destructor
    m_sim_data_nr = nullptr;
    m_sim_states_nr = nullptr;
    device_response_nr = nullptr;

    // Since hipMallocManaged does not call the constructor for non-POD types, we create cpu structs and fill them up
    // and then copy them over to the simData structs
    d18::VehicleParam veh_param;
    d18::TMeasyParam tire_param;

    setVehParamsJSON(veh_param, vehicle_params_file.c_str());
    setTireParamsJSON(tire_param, tire_params_file.c_str());
    // Initialize tire parameters that depend on other parameters
    tireInit(&tire_param);

    DriverData driver_data;
    LoadDriverData(driver_data, driver_inputs_file);
    unsigned int driver_data_len = driver_data.size();
    size_t old_vehicle_count = m_vehicle_count_tracker_params;
    m_vehicle_count_tracker_params += num_vehicles;
    for (size_t i = old_vehicle_count; i <= m_vehicle_count_tracker_params; i++) {
        m_sim_data[i]._driver_data_len = driver_data_len;
        // Allocate memory for the driver data
        CHECK_CUDA_ERROR(hipMallocManaged((void**)&m_sim_data[i]._driver_data,
                                           sizeof(DriverInput) * m_sim_data[i]._driver_data_len));
        // Copy the driver data from cpu to managed memory
        std::copy(driver_data.begin(), driver_data.end(), m_sim_data[i]._driver_data);
        // Fill up simulation data from the cpu structs
        m_sim_data[i]._veh_param = veh_param;
        m_sim_data[i]._tireTM_param = tire_param;
        // Set the final integration time for each of the vehicles
        m_sim_data[i]._t_end = driver_data.back().m_time;
    }
    hipMemPrefetchAsync(&m_sim_data, sizeof(SimData) * m_vehicle_count_tracker_params,
                         0);  // move the simData onto the GPU
}

__host__ void d18SolverHalfImplicitGPU::Construct(const std::string& vehicle_params_file,
                                                  const std::string& tire_params_file,
                                                  unsigned int num_vehicles,
                                                  const std::string& driver_inputs_file,
                                                  TireType type) {
    // Check if num_vehicles added is less than the total number of vehicles
    assert((num_vehicles + m_vehicle_count_tracker_params <= m_total_num_vehicles) &&
           "Number of vehicles added makes the vehicle count greater than the total number of vehicles");
    // If there is no tire type specified, then use TMeasy
    m_tire_type = type;

    if (m_tire_type == TireType::TMeasy) {
        // Because of this, we free the memory of the TMeasyNR tire
        hipFree(m_sim_data_nr);
        hipFree(m_sim_states_nr);
        hipFree(device_response_nr);
        // Set these to nullptr so that we don't try to free them again in the destructor
        m_sim_data_nr = nullptr;
        m_sim_states_nr = nullptr;
        device_response_nr = nullptr;

        // Since hipMallocManaged does not call the constructor for non-POD types, we create cpu structs and fill them
        // up and then copy them over to the simData structs
        d18::VehicleParam veh_param;
        d18::TMeasyParam tire_param;

        setVehParamsJSON(veh_param, vehicle_params_file.c_str());
        setTireParamsJSON(tire_param, tire_params_file.c_str());
        // Initialize tire parameters that depend on other parameters
        tireInit(&tire_param);

        DriverData driver_data;
        LoadDriverData(driver_data, driver_inputs_file);
        unsigned int driver_data_len = driver_data.size();
        size_t old_vehicle_count = m_vehicle_count_tracker_params;
        m_vehicle_count_tracker_params += num_vehicles;
        for (size_t i = old_vehicle_count; i <= m_vehicle_count_tracker_params; i++) {
            m_sim_data[i]._driver_data_len = driver_data_len;
            // Allocate memory for the driver data
            CHECK_CUDA_ERROR(hipMallocManaged((void**)&m_sim_data[i]._driver_data,
                                               sizeof(DriverInput) * m_sim_data[i]._driver_data_len));
            // Copy the driver data from cpu to managed memory
            std::copy(driver_data.begin(), driver_data.end(), m_sim_data[i]._driver_data);
            // Fill up simulation data from the cpu structs
            m_sim_data[i]._veh_param = veh_param;
            m_sim_data[i]._tireTM_param = tire_param;
            // Set the final integration time for each of the vehicles
            m_sim_data[i]._t_end = driver_data.back().m_time;
        }
        hipMemPrefetchAsync(&m_sim_data, sizeof(SimData) * m_vehicle_count_tracker_params,
                             0);  // move the simData onto the GPU
    } else {
        // Because of this, we free the memory of the TMeasyNR tire
        hipFree(m_sim_data);
        hipFree(m_sim_states);
        hipFree(device_response);
        // Set these to nullptr so that we don't try to free them again in the destructor
        m_sim_data = nullptr;
        m_sim_states = nullptr;
        device_response = nullptr;

        // Since hipMallocManaged does not call the constructor for non-POD types, we create cpu structs and fill them
        // up and then copy them over to the simData structs
        d18::VehicleParam veh_param;
        d18::TMeasyNrParam tire_param;

        setVehParamsJSON(veh_param, vehicle_params_file.c_str());
        setTireParamsJSON(tire_param, tire_params_file.c_str());
        // Initialize tire parameters that depend on other parameters
        tireInit(&tire_param);

        DriverData driver_data;
        LoadDriverData(driver_data, driver_inputs_file);
        unsigned int driver_data_len = driver_data.size();
        size_t old_vehicle_count = m_vehicle_count_tracker_params;
        m_vehicle_count_tracker_params += num_vehicles;
        for (size_t i = old_vehicle_count; i <= m_vehicle_count_tracker_params; i++) {
            m_sim_data_nr[i]._driver_data_len = driver_data_len;
            // Allocate memory for the driver data
            CHECK_CUDA_ERROR(hipMallocManaged((void**)&m_sim_data_nr[i]._driver_data,
                                               sizeof(DriverInput) * m_sim_data_nr[i]._driver_data_len));
            // Copy the driver data from cpu to managed memory
            std::copy(driver_data.begin(), driver_data.end(), m_sim_data_nr[i]._driver_data);
            // Fill up simulation data from the cpu structs
            m_sim_data_nr[i]._veh_param = veh_param;
            m_sim_data_nr[i]._tireTMNr_param = tire_param;
            // Set the final integration time for each of the vehicles
            m_sim_data_nr[i]._t_end = driver_data.back().m_time;
        }
        hipMemPrefetchAsync(&m_sim_data_nr, sizeof(SimData) * m_vehicle_count_tracker_params,
                             0);  // move the simData onto the GPU
    }
}

// Overload for situations when a controller is used and we don't have a driver data file
__host__ void d18SolverHalfImplicitGPU::Construct(const std::string& vehicle_params_file,
                                                  const std::string& tire_params_file,
                                                  unsigned int num_vehicles) {
    // Check if num_vehicles added is less than the total number of vehicles
    assert((num_vehicles + m_vehicle_count_tracker_params <= m_total_num_vehicles) &&
           "Number of vehicles added makes the vehicle count greater than the total number of vehicles");
    // If there is no tire type specified, then use TMeasy
    m_tire_type = TireType::TMeasy;
    // Because of this, we free the memory of the TMeasyNR tire
    hipFree(m_sim_data_nr);
    hipFree(m_sim_states_nr);
    hipFree(device_response_nr);
    // Set these to nullptr so that we don't try to free them again in the destructor
    m_sim_data_nr = nullptr;
    m_sim_states_nr = nullptr;
    device_response_nr = nullptr;

    // Since hipMallocManaged does not call the constructor for non-POD types, we create cpu structs and fill them up
    // and then copy them over to the simData structs
    d18::VehicleParam veh_param;
    d18::TMeasyParam tire_param;

    setVehParamsJSON(veh_param, vehicle_params_file.c_str());
    setTireParamsJSON(tire_param, tire_params_file.c_str());
    // Initialize tire parameters that depend on other parameters
    tireInit(&tire_param);

    size_t old_vehicle_count = m_vehicle_count_tracker_params;
    m_vehicle_count_tracker_params += num_vehicles;
    for (size_t i = old_vehicle_count; i <= m_vehicle_count_tracker_params; i++) {
        // Fill up simulation data from the cpu structs
        m_sim_data[i]._veh_param = veh_param;
        m_sim_data[i]._tireTM_param = tire_param;
    }

    hipMemPrefetchAsync(&m_sim_data, sizeof(SimData) * m_vehicle_count_tracker_params,
                         0);  // move the simData onto the GPU
}

__host__ void d18SolverHalfImplicitGPU::Construct(const std::string& vehicle_params_file,
                                                  const std::string& tire_params_file,
                                                  unsigned int num_vehicles,
                                                  TireType type) {
    // Check if num_vehicles added is less than the total number of vehicles
    assert((num_vehicles + m_vehicle_count_tracker_params <= m_total_num_vehicles) &&
           "Number of vehicles added makes the vehicle count greater than the total number of vehicles");
    m_tire_type = type;
    // If there is no tire type specified, then use TMeasy
    if (m_tire_type == TireType::TMeasy) {
        // Because of this, we free the memory of the TMeasyNR tire
        hipFree(m_sim_data_nr);
        hipFree(m_sim_states_nr);
        hipFree(device_response_nr);
        // Set these to nullptr so that we don't try to free them again in the destructor
        m_sim_data_nr = nullptr;
        m_sim_states_nr = nullptr;
        device_response_nr = nullptr;

        // Since hipMallocManaged does not call the constructor for non-POD types, we create cpu structs and fill them
        // up and then copy them over to the simData structs
        d18::VehicleParam veh_param;
        d18::TMeasyParam tire_param;

        setVehParamsJSON(veh_param, vehicle_params_file.c_str());
        setTireParamsJSON(tire_param, tire_params_file.c_str());
        // Initialize tire parameters that depend on other parameters
        tireInit(&tire_param);

        size_t old_vehicle_count = m_vehicle_count_tracker_params;
        m_vehicle_count_tracker_params += num_vehicles;
        for (size_t i = old_vehicle_count; i <= m_vehicle_count_tracker_params; i++) {
            // Fill up simulation data from the cpu structs
            m_sim_data[i]._veh_param = veh_param;
            m_sim_data[i]._tireTM_param = tire_param;
        }
        hipMemPrefetchAsync(&m_sim_data, sizeof(SimData) * m_vehicle_count_tracker_params,
                             0);  // move the simData onto the GPU
    } else {
        // Because of this, we free the memory of the TMeasyNR tire
        hipFree(m_sim_data);
        hipFree(m_sim_states);
        hipFree(device_response);
        // Set these to nullptr so that we don't try to free them again in the destructor
        m_sim_data = nullptr;
        m_sim_states = nullptr;
        device_response = nullptr;

        // Since hipMallocManaged does not call the constructor for non-POD types, we create cpu structs and fill them
        // up and then copy them over to the simData structs
        d18::VehicleParam veh_param;
        d18::TMeasyNrParam tire_param;

        setVehParamsJSON(veh_param, vehicle_params_file.c_str());
        setTireParamsJSON(tire_param, tire_params_file.c_str());
        // Initialize tire parameters that depend on other parameters
        tireInit(&tire_param);

        size_t old_vehicle_count = m_vehicle_count_tracker_params;
        m_vehicle_count_tracker_params += num_vehicles;
        for (size_t i = old_vehicle_count; i <= m_vehicle_count_tracker_params; i++) {
            // Fill up simulation data from the cpu structs
            m_sim_data_nr[i]._veh_param = veh_param;
            m_sim_data_nr[i]._tireTMNr_param = tire_param;
        }
        hipMemPrefetchAsync(&m_sim_data_nr, sizeof(SimData) * m_vehicle_count_tracker_params,
                             0);  // move the simData onto the GPU
    }
}
// ======================================================================================================================

__host__ void d18SolverHalfImplicitGPU::Initialize(d18::VehicleState& vehicle_states,
                                                   d18::TMeasyState& tire_states_LF,
                                                   d18::TMeasyState& tire_states_RF,
                                                   d18::TMeasyState& tire_states_LR,
                                                   d18::TMeasyState& tire_states_RR,
                                                   unsigned int num_vehicles) {
    // Esnure that construct was called with TMeasy tire type
    assert((m_tire_type == TireType::TMeasy) &&
           "Construct function called with TMeasyNr tire type, but Initialize called with TMeasy tire type");

    size_t old_vehicle_count = m_vehicle_count_tracker_states;
    m_vehicle_count_tracker_states += num_vehicles;
    for (size_t i = old_vehicle_count; i <= m_vehicle_count_tracker_states; i++) {
        // Fill up simulation data from the cpu structs
        m_sim_states[i]._veh_state = vehicle_states;
        m_sim_states[i]._tirelf_state = tire_states_LF;
        m_sim_states[i]._tirerf_state = tire_states_RF;
        m_sim_states[i]._tirelr_state = tire_states_LR;
        m_sim_states[i]._tirerr_state = tire_states_RR;
    }
    hipMemPrefetchAsync(&m_sim_states, sizeof(SimState) * m_vehicle_count_tracker_states,
                         0);  // move the simState onto the GPU

    // TODO: Add Jacobian support
    // // Size the jacobian matrices - size relies on the torque converter bool
    // m_num_controls = 2;
    // if (m_veh_param._tcbool) {
    //     m_num_states = 21;
    //     m_jacobian_state.resize(m_num_states, std::vector<double>(m_num_states, 0));
    //     m_jacobian_controls.resize(m_num_states, std::vector<double>(m_num_controls, 0));
    // } else {
    //     m_num_states = 20;
    //     m_jacobian_state.resize(m_num_states, std::vector<double>(m_num_states, 0));
    //     m_jacobian_controls.resize(m_num_states, std::vector<double>(m_num_controls, 0));
    // }
}

// TMeasy without relaxation does not have tire states and so the jacobian size reduces by 8
__host__ void d18SolverHalfImplicitGPU::Initialize(d18::VehicleState& vehicle_states,
                                                   d18::TMeasyNrState& tire_states_LF,
                                                   d18::TMeasyNrState& tire_states_RF,
                                                   d18::TMeasyNrState& tire_states_LR,
                                                   d18::TMeasyNrState& tire_states_RR,
                                                   unsigned int num_vehicles) {
    // Esnure that construct was called with TMeasyNr tire type
    assert((m_tire_type == TireType::TMeasyNr) &&
           "Construct function called with TMeasy tire type, but Initialize called with TMeasyNR tire type");
    size_t old_vehicle_count = m_vehicle_count_tracker_states;
    m_vehicle_count_tracker_states += num_vehicles;
    for (size_t i = old_vehicle_count; i <= m_vehicle_count_tracker_states; i++) {
        // Fill up simulation data from the cpu structs
        m_sim_states_nr[i]._veh_state = vehicle_states;
        m_sim_states_nr[i]._tirelf_state = tire_states_LF;
        m_sim_states_nr[i]._tirerf_state = tire_states_RF;
        m_sim_states_nr[i]._tirelr_state = tire_states_LR;
        m_sim_states_nr[i]._tirerr_state = tire_states_RR;
    }
    hipMemPrefetchAsync(&m_sim_states_nr, sizeof(SimState) * m_vehicle_count_tracker_states,
                         0);  // move the simState onto the GPU

    // TODO: Add Jacobian support
    // Size the jacobian matrices - size relies on the torque converter bool
    // m_num_controls = 2;
    // if (m_veh_param._tcbool) {
    //     m_num_states = 13;
    //     m_jacobian_state.resize(m_num_states, std::vector<double>(m_num_states, 0));
    //     m_jacobian_controls.resize(m_num_states, std::vector<double>(m_num_controls, 0));
    // } else {
    //     m_num_states = 12;
    //     m_jacobian_state.resize(m_num_states, std::vector<double>(m_num_states, 0));
    //     m_jacobian_controls.resize(m_num_states, std::vector<double>(m_num_controls, 0));
    // }
}

// ======================================================================================================================

/// @brief Sets the path for the output file
/// @param output_file string with full path with extension
void d18SolverHalfImplicitGPU::SetOutput(const std::string& output_file, double output_freq) {
    m_output = true;
    m_output_file = output_file;
    m_timeStepsStored = 0;
    m_dtout = 1.0 / output_freq;
}

// ======================================================================================================================

/// @brief Solve the system of equations by calling the integrate function
void d18SolverHalfImplicitGPU::Solve() {
    assert(!m_driver_data.empty() && "No controls provided, please use construct to pass path to driver inputs");

    // For now just integrate to final time
    Integrate();
}

// ======================================================================================================================

/// @brief Integrate the system of equations using the half implicit method - Calls the RHS function at each time step
void d18SolverHalfImplicitGPU::Integrate() {
    double t = 0;
    // Create output writer
    if (m_output) {
        Write(t);
        m_timeStepsStored++;
    }

    // Integrate to final time by repeatedly calling the RHS function
    while (t < (m_tend - m_step / 10.)) {
        // Call RHS to get all accelerations
        rhsFun(t);

        // Integrate according to half implicit method for second order states
        // Integrate according to explicit method for first order states

        if (m_tire_type == TireType::TMeasy) {  // Only TM easy has xe and ye states
            // First the tire states
            // LF
            m_tireTMlf_state._xe += m_tireTMlf_state._xedot * m_step;
            m_tireTMlf_state._ye += m_tireTMlf_state._yedot * m_step;
            m_tireTMlf_state._omega += m_tireTMlf_state._dOmega * m_step;
            // RF
            m_tireTMrf_state._xe += m_tireTMrf_state._xedot * m_step;
            m_tireTMrf_state._ye += m_tireTMrf_state._yedot * m_step;
            m_tireTMrf_state._omega += m_tireTMrf_state._dOmega * m_step;
            // LR
            m_tireTMlr_state._xe += m_tireTMlr_state._xedot * m_step;
            m_tireTMlr_state._ye += m_tireTMlr_state._yedot * m_step;
            m_tireTMlr_state._omega += m_tireTMlr_state._dOmega * m_step;
            // RR
            m_tireTMrr_state._xe += m_tireTMrr_state._xedot * m_step;
            m_tireTMrr_state._ye += m_tireTMrr_state._yedot * m_step;
            m_tireTMrr_state._omega += m_tireTMrr_state._dOmega * m_step;
        } else {  // Other tires have only omega states
            // First the tire states
            // LF
            m_tireTMNrlf_state._omega += m_tireTMNrlf_state._dOmega * m_step;
            // RF
            m_tireTMNrrf_state._omega += m_tireTMNrrf_state._dOmega * m_step;
            // LR
            m_tireTMNrlr_state._omega += m_tireTMNrlr_state._dOmega * m_step;
            // RR
            m_tireTMNrrr_state._omega += m_tireTMNrrr_state._dOmega * m_step;
        }

        // Now the vehicle states
        if (m_veh_param._tcbool) {
            m_veh_state._crankOmega += m_veh_state._dOmega_crank * m_step;
        }

        // Integrate velocity level first
        m_veh_state._u += m_veh_state._udot * m_step;
        m_veh_state._v += m_veh_state._vdot * m_step;
        m_veh_state._wx += m_veh_state._wxdot * m_step;
        m_veh_state._wz += m_veh_state._wzdot * m_step;

        // Integrate position level next
        m_veh_state._x +=
            (m_veh_state._u * std::cos(m_veh_state._psi) - m_veh_state._v * std::sin(m_veh_state._psi)) * m_step;
        m_veh_state._y +=
            (m_veh_state._u * std::sin(m_veh_state._psi) + m_veh_state._v * std::cos(m_veh_state._psi)) * m_step;
        m_veh_state._psi += m_veh_state._wz * m_step;
        m_veh_state._phi += m_veh_state._wx * m_step;

        // Update time
        t += m_step;

        // Write the output
        if (m_output) {
            if (std::abs(t - m_timeStepsStored * m_dtout) < 1e-7) {
                Write(t);
                m_timeStepsStored++;
            }
        }
    }
    if (m_output) {
        WriteToFile();
    }
}
// ======================================================================================================================

/// @brief Function call to integrate by just a single time step. This function will always integrate to the t + m_step
/// where m_step is set using the SetTimeStep function.
/// @param t current time
/// @param throttle throttle input
/// @param steering steering input
/// @param braking braking input
/// @return t + m_step
double d18SolverHalfImplicitGPU::IntegrateStep(double t, double throttle, double steering, double braking) {
    // Store header and first time step
    if (m_output && (t < m_step)) {
        Write(t);
        m_timeStepsStored++;
    }

    DriverInput controls(t, steering, throttle, braking);
    // Call the RHS function
    rhsFun(t, controls);

    // Integrate according to half implicit method for second order states
    // Integrate according to explicit method for first order states

    if (m_tire_type == TireType::TMeasy) {  // Only TM easy has xe and ye states
        // First the tire states
        // LF
        m_tireTMlf_state._xe += m_tireTMlf_state._xedot * m_step;
        m_tireTMlf_state._ye += m_tireTMlf_state._yedot * m_step;
        m_tireTMlf_state._omega += m_tireTMlf_state._dOmega * m_step;
        // RF
        m_tireTMrf_state._xe += m_tireTMrf_state._xedot * m_step;
        m_tireTMrf_state._ye += m_tireTMrf_state._yedot * m_step;
        m_tireTMrf_state._omega += m_tireTMrf_state._dOmega * m_step;
        // LR
        m_tireTMlr_state._xe += m_tireTMlr_state._xedot * m_step;
        m_tireTMlr_state._ye += m_tireTMlr_state._yedot * m_step;
        m_tireTMlr_state._omega += m_tireTMlr_state._dOmega * m_step;
        // RR
        m_tireTMrr_state._xe += m_tireTMrr_state._xedot * m_step;
        m_tireTMrr_state._ye += m_tireTMrr_state._yedot * m_step;
        m_tireTMrr_state._omega += m_tireTMrr_state._dOmega * m_step;
    } else {  // Other tires have only omega states
        // First the tire states
        // LF
        m_tireTMNrlf_state._omega += m_tireTMNrlf_state._dOmega * m_step;
        // RF
        m_tireTMNrrf_state._omega += m_tireTMNrrf_state._dOmega * m_step;
        // LR
        m_tireTMNrlr_state._omega += m_tireTMNrlr_state._dOmega * m_step;
        // RR
        m_tireTMNrrr_state._omega += m_tireTMNrrr_state._dOmega * m_step;
    }

    // Now the vehicle states
    if (m_veh_param._tcbool) {
        m_veh_state._crankOmega += m_veh_state._dOmega_crank * m_step;
    }

    // Integrate velocity level first
    m_veh_state._u += m_veh_state._udot * m_step;
    m_veh_state._v += m_veh_state._vdot * m_step;
    m_veh_state._wx += m_veh_state._wxdot * m_step;
    m_veh_state._wz += m_veh_state._wzdot * m_step;

    // Integrate position level next
    m_veh_state._x +=
        (m_veh_state._u * std::cos(m_veh_state._psi) - m_veh_state._v * std::sin(m_veh_state._psi)) * m_step;
    m_veh_state._y +=
        (m_veh_state._u * std::sin(m_veh_state._psi) + m_veh_state._v * std::cos(m_veh_state._psi)) * m_step;
    m_veh_state._psi += m_veh_state._wz * m_step;
    m_veh_state._phi += m_veh_state._wx * m_step;

    double new_time = t + m_step;
    // Write the output
    if (m_output) {
        if (std::abs(new_time - m_timeStepsStored * m_dtout) < 1e-7) {
            Write(new_time);
            m_timeStepsStored++;
        }
    }

    return new_time;
}
// ======================================================================================================================
/// @brief Function call to integrate by just a single time step with jacobian computation. The order of the states in
// the jacobian matrix if the TMeasy tire is used are is as follows:
//  0: tirelf_st._xe;
//  1: tirelf_st._ye;
//  2: tirerf_st._xe;
//  3: tirerf_st._ye;
//  4: tirelr_st._xe;
//  5: tirelr_st._ye;
//  6: tirerr_st._xe;
//  7: tirerr_st._ye;
//  8: tirelf_st._omega;
//  9: tirerf_st._omega;
//  10: tirelr_st._omega;
//  11: tirerr_st._omega;
//  12: v_states._crankOmega; (only if torque converter is used)
//  13: v_states._x;
//  14: v_states._y;
//  15: v_states._u;
//  16: v_states._v;
//  17: v_states._psi;
//  18: v_states._wz;
//  19: v_states._phi;
//  20: v_states._wx;
// If the TMeasyNR tire is used then the order of the states in the jacobian matrix are as follows:
//  0: tirelf_st._omega;
//  1: tirerf_st._omega;
//  2: tirelr_st._omega;
//  3: tirerr_st._omega;
//  4: v_states._crankOmega; (only if torque converter is used)
//  5: v_states._x;
//  6: v_states._y;
//  7: v_states._u;
//  8: v_states._v;
//  9: v_states._psi;
//  10: v_states._wz;
//  11: v_states._phi;
//  12: v_states._wx;
/// @param t current time
/// @param throttle throttle input
/// @param steering steering input
/// @param braking braking input
/// @param on boolean to turn on jacobian computation
/// @return t + m_step
double d18SolverHalfImplicitGPU::IntegrateStepWithJacobian(double t,
                                                           double throttle,
                                                           double steering,
                                                           double braking,
                                                           bool jacOn) {
    // Store header and first time step
    if (m_output && (t < m_step)) {
        Write(t);
        m_timeStepsStored++;
    }

    DriverInput controls(t, steering, throttle, braking);

    // If the jacobian switch is on, then compute the jacobian
    if (jacOn) {
        std::vector<double> y(m_num_states, 0);
        std::vector<double> ydot(m_num_states, 0);
        // package all the current states
        if (m_tire_type == TireType::TMeasy) {
            packY(m_veh_state, m_tireTMlf_state, m_tireTMrf_state, m_tireTMlr_state, m_tireTMrr_state,
                  m_veh_param._tcbool, y);
        } else {
            packY(m_veh_state, m_tireTMNrlf_state, m_tireTMNrrf_state, m_tireTMNrlr_state, m_tireTMNrrr_state,
                  m_veh_param._tcbool, y);
        }

        // ============================
        // Computing the state jacobian
        // ============================

        // Set a vector of del Ys - for now set this to some scale of y
        std::vector<double> delY(y.begin(), y.end());
        // In a loop pertub each state and get the corresponding perturbed ydot
        int ySize = y.size();

#pragma omp parallel for simd
        for (int i = 0; i < ySize; i++) {
            // Perterbation is 1e-3 * y (since some states are really small values wile some are huge)
            delY[i] = std::abs(delY[i] * 1e-3);
            if (delY[i] < 1e-8) {
                // This means that the particular state is 0. In this case set dels to 1e-3
                delY[i] = 1e-3;
            }

            // perturb y at i
            std::vector<double> perturbedYPlus(y.begin(), y.end());
            std::vector<double> perturbedYMinus(y.begin(), y.end());

            perturbedYPlus[i] = perturbedYPlus[i] + delY[i];
            perturbedYMinus[i] = perturbedYMinus[i] - delY[i];

            // ydots to store the output
            std::vector<double> ydotPlus(y.size(), 0.);
            std::vector<double> ydotMinus(y.size(), 0.);

            // Call the perturb function with these to get the perturbed ydot -> This does not update the state
            PerturbRhsFun(perturbedYPlus, controls, ydotPlus);
            PerturbRhsFun(perturbedYMinus, controls, ydotMinus);

// Update the jacobian matrix
#pragma omp simd
            for (int j = 0; j < ySize; j++) {
                m_jacobian_state[j][i] = (ydotPlus[j] - ydotMinus[j]) / (2 * delY[i]);
            }
        }

        // ==============================
        // Computing the control jacobian
        //===============================

        // Set a vector of del controls - for now we ingnore braking
        std::vector<double> delControls = {1e-3, 1e-3};
        // In a loop pertub each control and get the corresponding perturbed ydot
        int controlSize = delControls.size();

        for (int i = 0; i < controlSize; i++) {
            // perturb controls at i
            std::vector<double> perturbedControlsPlus = {steering, throttle};
            std::vector<double> perturbedControlsMinus = {steering, throttle};

            perturbedControlsPlus[i] = perturbedControlsPlus[i] + delControls[i];
            perturbedControlsMinus[i] = perturbedControlsMinus[i] - delControls[i];

            // ydots to store the output
            std::vector<double> ydotPlus(y.size(), 0.);
            std::vector<double> ydotMinus(y.size(), 0.);

            // Call the perturb function with these to get the perturbed ydot -> This does not update the state
            DriverInput inputPlus(t, perturbedControlsPlus[0], perturbedControlsPlus[1], 0);
            DriverInput inputMinus(t, perturbedControlsMinus[0], perturbedControlsMinus[1], 0);
            PerturbRhsFun(y, inputPlus, ydotPlus);
            PerturbRhsFun(y, inputMinus, ydotMinus);

            // Update the jacobian matrix
            for (int j = 0; j < ySize; j++) {
                m_jacobian_controls[j][i] = (ydotPlus[j] - ydotMinus[j]) / (2 * delControls[i]);
            }
        }
    }

    // Call the RHS function
    rhsFun(t, controls);

    // Integrate according to half implicit method for second order states
    // Integrate according to explicit method for first order states

    if (m_tire_type == TireType::TMeasy) {  // Only TM easy has xe and ye states
        // First the tire states
        // LF
        m_tireTMlf_state._xe += m_tireTMlf_state._xedot * m_step;
        m_tireTMlf_state._ye += m_tireTMlf_state._yedot * m_step;
        m_tireTMlf_state._omega += m_tireTMlf_state._dOmega * m_step;
        // RF
        m_tireTMrf_state._xe += m_tireTMrf_state._xedot * m_step;
        m_tireTMrf_state._ye += m_tireTMrf_state._yedot * m_step;
        m_tireTMrf_state._omega += m_tireTMrf_state._dOmega * m_step;
        // LR
        m_tireTMlr_state._xe += m_tireTMlr_state._xedot * m_step;
        m_tireTMlr_state._ye += m_tireTMlr_state._yedot * m_step;
        m_tireTMlr_state._omega += m_tireTMlr_state._dOmega * m_step;
        // RR
        m_tireTMrr_state._xe += m_tireTMrr_state._xedot * m_step;
        m_tireTMrr_state._ye += m_tireTMrr_state._yedot * m_step;
        m_tireTMrr_state._omega += m_tireTMrr_state._dOmega * m_step;
    } else {  // Other tires have only omega states
        // First the tire states
        // LF
        m_tireTMNrlf_state._omega += m_tireTMNrlf_state._dOmega * m_step;
        // RF
        m_tireTMNrrf_state._omega += m_tireTMNrrf_state._dOmega * m_step;
        // LR
        m_tireTMNrlr_state._omega += m_tireTMNrlr_state._dOmega * m_step;
        // RR
        m_tireTMNrrr_state._omega += m_tireTMNrrr_state._dOmega * m_step;
    }
    // Now the vehicle states
    if (m_veh_param._tcbool) {
        m_veh_state._crankOmega += m_veh_state._dOmega_crank * m_step;
    }

    // Integrate velocity level first
    m_veh_state._u += m_veh_state._udot * m_step;
    m_veh_state._v += m_veh_state._vdot * m_step;
    m_veh_state._wx += m_veh_state._wxdot * m_step;
    m_veh_state._wz += m_veh_state._wzdot * m_step;

    // Integrate position level next
    m_veh_state._x +=
        (m_veh_state._u * std::cos(m_veh_state._psi) - m_veh_state._v * std::sin(m_veh_state._psi)) * m_step;
    m_veh_state._y +=
        (m_veh_state._u * std::sin(m_veh_state._psi) + m_veh_state._v * std::cos(m_veh_state._psi)) * m_step;
    m_veh_state._psi += m_veh_state._wz * m_step;
    m_veh_state._phi += m_veh_state._wx * m_step;

    double new_time = t + m_step;
    // Write the output
    if (m_output) {
        if (std::abs(new_time - m_timeStepsStored * m_dtout) < 1e-7) {
            Write(new_time);
            m_timeStepsStored++;
        }
    }

    return new_time;
}

// ======================================================================================================================

/// @brief Computes the RHS of all the ODEs (tire velocities, chassis accelerations)
/// @param t Current time
void d18SolverHalfImplicitGPU::rhsFun(double t) {
    // Get controls at the current timeStep
    auto controls = GetDriverInput(t, m_driver_data);

    // Calculate tire vertical loads
    std::vector<double> loads(4, 0);
    if (m_tire_type == TireType::TMeasy) {
        computeTireLoads(loads, m_veh_state, m_veh_param, m_tireTM_param);

        // Transform from vehicle frame to the tire frame
        vehToTireTransform(m_tireTMlf_state, m_tireTMrf_state, m_tireTMlr_state, m_tireTMrr_state, m_veh_state, loads,
                           m_veh_param, controls.m_steering);

        // Tire velocities using TMEasy tire
        computeTireRHS(m_tireTMlf_state, m_tireTM_param, m_veh_param, controls.m_steering);
        computeTireRHS(m_tireTMrf_state, m_tireTM_param, m_veh_param, controls.m_steering);
        computeTireRHS(m_tireTMlr_state, m_tireTM_param, m_veh_param, 0);  // No rear steering
        computeTireRHS(m_tireTMrr_state, m_tireTM_param, m_veh_param, 0);  // No rear steering

        // Powertrain dynamics
        computePowertrainRHS(m_veh_state, m_tireTMlf_state, m_tireTMrf_state, m_tireTMlr_state, m_tireTMrr_state,
                             m_veh_param, m_tireTM_param, controls);
//////// DEBUG
#ifdef DEBUG
        M_DEBUG_LF_TIRE_FX = m_tireTMlf_state._fx;
        M_DEBUG_RF_TIRE_FX = m_tireTMrf_state._fx;
        M_DEBUG_LR_TIRE_FX = m_tireTMlr_state._fx;
        M_DEBUG_RR_TIRE_FX = m_tireTMrr_state._fx;

        M_DEBUG_LF_TIRE_FY = m_tireTMlf_state._fy;
        M_DEBUG_RF_TIRE_FY = m_tireTMrf_state._fy;
        M_DEBUG_LR_TIRE_FY = m_tireTMlr_state._fy;
        M_DEBUG_RR_TIRE_FY = m_tireTMrr_state._fy;

        M_DEBUG_LF_TIRE_FZ = m_tireTMlf_state._fz;
        M_DEBUG_RF_TIRE_FZ = m_tireTMrf_state._fz;
        M_DEBUG_LR_TIRE_FZ = m_tireTMlr_state._fz;
        M_DEBUG_RR_TIRE_FZ = m_tireTMrr_state._fz;
#endif

        // Vehicle dynamics
        tireToVehTransform(m_tireTMlf_state, m_tireTMrf_state, m_tireTMlr_state, m_tireTMrr_state, m_veh_state,
                           m_veh_param, controls.m_steering);
        std::vector<double> fx = {m_tireTMlf_state._fx, m_tireTMrf_state._fx, m_tireTMlr_state._fx,
                                  m_tireTMrr_state._fx};
        std::vector<double> fy = {m_tireTMlf_state._fy, m_tireTMrf_state._fy, m_tireTMlr_state._fy,
                                  m_tireTMrr_state._fy};
        computeVehRHS(m_veh_state, m_veh_param, fx, fy);
    } else {  // For the other tire
        computeTireLoads(loads, m_veh_state, m_veh_param, m_tireTMNr_param);

        // Transform from vehicle frame to the tire frame
        vehToTireTransform(m_tireTMNrlf_state, m_tireTMNrrf_state, m_tireTMNrlr_state, m_tireTMNrrr_state, m_veh_state,
                           loads, m_veh_param, controls.m_steering);

        // Tire velocities using TMEasy tire
        computeTireRHS(m_tireTMNrlf_state, m_tireTMNr_param, m_veh_param, controls.m_steering);
        computeTireRHS(m_tireTMNrrf_state, m_tireTMNr_param, m_veh_param, controls.m_steering);
        computeTireRHS(m_tireTMNrlr_state, m_tireTMNr_param, m_veh_param, 0);  // No rear steering
        computeTireRHS(m_tireTMNrrr_state, m_tireTMNr_param, m_veh_param, 0);  // No rear steering

        // Powertrain dynamics
        computePowertrainRHS(m_veh_state, m_tireTMNrlf_state, m_tireTMNrrf_state, m_tireTMNrlr_state,
                             m_tireTMNrrr_state, m_veh_param, m_tireTMNr_param, controls);
//////// DEBUG
#ifdef DEBUG
        M_DEBUG_LF_TIRE_FX = m_tireTMNrlf_state._fx;
        M_DEBUG_RF_TIRE_FX = m_tireTMNrrf_state._fx;
        M_DEBUG_LR_TIRE_FX = m_tireTMNrlr_state._fx;
        M_DEBUG_RR_TIRE_FX = m_tireTMNrrr_state._fx;

        M_DEBUG_LF_TIRE_FY = m_tireTMNrlf_state._fy;
        M_DEBUG_RF_TIRE_FY = m_tireTMNrrf_state._fy;
        M_DEBUG_LR_TIRE_FY = m_tireTMNrlr_state._fy;
        M_DEBUG_RR_TIRE_FY = m_tireTMNrrr_state._fy;

        M_DEBUG_LF_TIRE_FZ = m_tireTMNrlf_state._fz;
        M_DEBUG_RF_TIRE_FZ = m_tireTMNrrf_state._fz;
        M_DEBUG_LR_TIRE_FZ = m_tireTMNrlr_state._fz;
        M_DEBUG_RR_TIRE_FZ = m_tireTMNrrr_state._fz;
#endif

        // Vehicle dynamics
        tireToVehTransform(m_tireTMNrlf_state, m_tireTMNrrf_state, m_tireTMNrlr_state, m_tireTMNrrr_state, m_veh_state,
                           m_veh_param, controls.m_steering);
        std::vector<double> fx = {m_tireTMNrlf_state._fx, m_tireTMNrrf_state._fx, m_tireTMNrlr_state._fx,
                                  m_tireTMNrrr_state._fx};
        std::vector<double> fy = {m_tireTMNrlf_state._fy, m_tireTMNrrf_state._fy, m_tireTMNrlr_state._fy,
                                  m_tireTMNrrr_state._fy};
        computeVehRHS(m_veh_state, m_veh_param, fx, fy);
    }
}

// ======================================================================================================================

void d18SolverHalfImplicitGPU::rhsFun(double t, DriverInput& controls) {
    // Calculate tire vertical loads
    std::vector<double> loads(4, 0);
    if (m_tire_type == TireType::TMeasy) {
        computeTireLoads(loads, m_veh_state, m_veh_param, m_tireTM_param);

        // Transform from vehicle frame to the tire frame
        vehToTireTransform(m_tireTMlf_state, m_tireTMrf_state, m_tireTMlr_state, m_tireTMrr_state, m_veh_state, loads,
                           m_veh_param, controls.m_steering);

        // Tire velocities using TMEasy tire
        computeTireRHS(m_tireTMlf_state, m_tireTM_param, m_veh_param, controls.m_steering);
        computeTireRHS(m_tireTMrf_state, m_tireTM_param, m_veh_param, controls.m_steering);
        computeTireRHS(m_tireTMlr_state, m_tireTM_param, m_veh_param, 0);  // No rear steering
        computeTireRHS(m_tireTMrr_state, m_tireTM_param, m_veh_param, 0);  // No rear steering

        // Powertrain dynamics
        computePowertrainRHS(m_veh_state, m_tireTMlf_state, m_tireTMrf_state, m_tireTMlr_state, m_tireTMrr_state,
                             m_veh_param, m_tireTM_param, controls);

        // Vehicle dynamics
        tireToVehTransform(m_tireTMlf_state, m_tireTMrf_state, m_tireTMlr_state, m_tireTMrr_state, m_veh_state,
                           m_veh_param, controls.m_steering);
        std::vector<double> fx = {m_tireTMlf_state._fx, m_tireTMrf_state._fx, m_tireTMlr_state._fx,
                                  m_tireTMrr_state._fx};
        std::vector<double> fy = {m_tireTMlf_state._fy, m_tireTMrf_state._fy, m_tireTMlr_state._fy,
                                  m_tireTMrr_state._fy};
        computeVehRHS(m_veh_state, m_veh_param, fx, fy);
    } else {  // For the other tire
        computeTireLoads(loads, m_veh_state, m_veh_param, m_tireTMNr_param);

        // Transform from vehicle frame to the tire frame
        vehToTireTransform(m_tireTMNrlf_state, m_tireTMNrrf_state, m_tireTMNrlr_state, m_tireTMNrrr_state, m_veh_state,
                           loads, m_veh_param, controls.m_steering);

        // Tire velocities using TMEasy tire
        computeTireRHS(m_tireTMNrlf_state, m_tireTMNr_param, m_veh_param, controls.m_steering);
        computeTireRHS(m_tireTMNrrf_state, m_tireTMNr_param, m_veh_param, controls.m_steering);
        computeTireRHS(m_tireTMNrlr_state, m_tireTMNr_param, m_veh_param, 0);  // No rear steering
        computeTireRHS(m_tireTMNrrr_state, m_tireTMNr_param, m_veh_param, 0);  // No rear steering

        // Powertrain dynamics
        computePowertrainRHS(m_veh_state, m_tireTMNrlf_state, m_tireTMNrrf_state, m_tireTMNrlr_state,
                             m_tireTMNrrr_state, m_veh_param, m_tireTMNr_param, controls);

        // Vehicle dynamics
        tireToVehTransform(m_tireTMNrlf_state, m_tireTMNrrf_state, m_tireTMNrlr_state, m_tireTMNrrr_state, m_veh_state,
                           m_veh_param, controls.m_steering);
        std::vector<double> fx = {m_tireTMNrlf_state._fx, m_tireTMNrrf_state._fx, m_tireTMNrlr_state._fx,
                                  m_tireTMNrrr_state._fx};
        std::vector<double> fy = {m_tireTMNrlf_state._fy, m_tireTMNrrf_state._fy, m_tireTMNrlr_state._fy,
                                  m_tireTMNrrr_state._fy};
        computeVehRHS(m_veh_state, m_veh_param, fx, fy);
    }
}

// ======================================================================================================================

// Function takes (y +- dely) and provides a new ydot for the pertubed y (ydot is the rhs of the system of equations)

void d18SolverHalfImplicitGPU::PerturbRhsFun(std::vector<double>& y, DriverInput& controls, std::vector<double>& ydot) {
    // Extract the vehicle and tire states vector state
    VehicleState veh_st;
    if (m_tire_type == TireType::TMeasy) {
        TMeasyState tirelf_st;
        TMeasyState tirerf_st;
        TMeasyState tirelr_st;
        TMeasyState tirerr_st;
        unpackY(y, m_veh_param._tcbool, veh_st, tirelf_st, tirerf_st, tirelr_st, tirerr_st);

        // Calculate tire vertical loads
        std::vector<double> loads(4, 0);
        computeTireLoads(loads, veh_st, m_veh_param, m_tireTM_param);

        // Transform from the vehicle frame to the tire frame
        vehToTireTransform(tirelf_st, tirerf_st, tirelr_st, tirerr_st, veh_st, loads, m_veh_param, controls.m_steering);

        // Tire dynamics
        computeTireRHS(tirelf_st, m_tireTM_param, m_veh_param, controls.m_steering);
        computeTireRHS(tirerf_st, m_tireTM_param, m_veh_param, controls.m_steering);
        computeTireRHS(tirelr_st, m_tireTM_param, m_veh_param, 0);
        computeTireRHS(tirerr_st, m_tireTM_param, m_veh_param, 0);

        // Powertrain dynamics
        computePowertrainRHS(veh_st, tirelf_st, tirerf_st, tirelr_st, tirerr_st, m_veh_param, m_tireTM_param, controls);

        // Vehicle dynamics
        tireToVehTransform(tirelf_st, tirerf_st, tirelr_st, tirerr_st, veh_st, m_veh_param, controls.m_steering);
        std::vector<double> fx = {tirelf_st._fx, tirerf_st._fx, tirelr_st._fx, tirerr_st._fx};
        std::vector<double> fy = {tirelf_st._fy, tirerf_st._fy, tirelr_st._fy, tirerr_st._fy};
        computeVehRHS(veh_st, m_veh_param, fx, fy);

        // Pack the ydot and send it
        packYDOT(veh_st, tirelf_st, tirerf_st, tirelr_st, tirerr_st, m_veh_param._tcbool, ydot);
    } else {
        TMeasyNrState tirelf_st;
        TMeasyNrState tirerf_st;
        TMeasyNrState tirelr_st;
        TMeasyNrState tirerr_st;
        unpackY(y, m_veh_param._tcbool, veh_st, tirelf_st, tirerf_st, tirelr_st, tirerr_st);

        // Calculate tire vertical loads
        std::vector<double> loads(4, 0);
        computeTireLoads(loads, veh_st, m_veh_param, m_tireTMNr_param);

        // Transform from the vehicle frame to the tire frame
        vehToTireTransform(tirelf_st, tirerf_st, tirelr_st, tirerr_st, veh_st, loads, m_veh_param, controls.m_steering);

        // Tire dynamics
        computeTireRHS(tirelf_st, m_tireTMNr_param, m_veh_param, controls.m_steering);
        computeTireRHS(tirerf_st, m_tireTMNr_param, m_veh_param, controls.m_steering);
        computeTireRHS(tirelr_st, m_tireTMNr_param, m_veh_param, 0);
        computeTireRHS(tirerr_st, m_tireTMNr_param, m_veh_param, 0);

        // Powertrain dynamics
        computePowertrainRHS(veh_st, tirelf_st, tirerf_st, tirelr_st, tirerr_st, m_veh_param, m_tireTMNr_param,
                             controls);

        // Vehicle dynamics
        tireToVehTransform(tirelf_st, tirerf_st, tirelr_st, tirerr_st, veh_st, m_veh_param, controls.m_steering);
        std::vector<double> fx = {tirelf_st._fx, tirerf_st._fx, tirelr_st._fx, tirerr_st._fx};
        std::vector<double> fy = {tirelf_st._fy, tirerf_st._fy, tirelr_st._fy, tirerr_st._fy};
        computeVehRHS(veh_st, m_veh_param, fx, fy);

        // Pack the ydot and send it
        packYDOT(veh_st, tirelf_st, tirerf_st, tirelr_st, tirerr_st, m_veh_param._tcbool, ydot);
    }
}

void d18SolverHalfImplicitGPU::Write(double t) {
    // If we are in initial time step, write the header
    if (t < m_step) {
        m_csv << "time";
        m_csv << "x";
        m_csv << "y";
        m_csv << "vx";
        m_csv << "vy";
        m_csv << "ax";
        m_csv << "ay";
        m_csv << "roll";
        m_csv << "yaw";
        m_csv << "roll_rate";
        m_csv << "yaw_rate";
        m_csv << "wlf";
        m_csv << "wrf";
        m_csv << "wlr";
        m_csv << "wrr";
        m_csv << "sp_tor";
        m_csv << "current_gear";
        m_csv << "engine_omega";
#ifdef DEBUG
        m_csv << "lf_tireForce_x";
        m_csv << "rf_tireForce_x";
        m_csv << "lr_tireForce_x";
        m_csv << "rr_tireForce_x";
        m_csv << "lf_tireForce_y";
        m_csv << "rf_tireForce_y";
        m_csv << "lr_tireForce_y";
        m_csv << "rr_tireForce_y";
        m_csv << "lf_tireForce_z";
        m_csv << "rf_tireForce_z";
        m_csv << "lr_tireForce_z";
        m_csv << "rr_tireForce_z";
#endif
        m_csv << std::endl;

        m_csv << 0;
        m_csv << 0;
        m_csv << 0;
        m_csv << 0;
        m_csv << 0;
        m_csv << 0;
        m_csv << 0;
        m_csv << 0;
        m_csv << 0;
        m_csv << 0;
        m_csv << 0;
        m_csv << 0;
        m_csv << 0;
        m_csv << 0;
        m_csv << 0;
        m_csv << 0;
        m_csv << 0;
        m_csv << 0;
#ifdef DEBUG
        m_csv << 0;
        m_csv << 0;
        m_csv << 0;
        m_csv << 0;
        m_csv << 0;
        m_csv << 0;
        m_csv << 0;
        m_csv << 0;
        m_csv << 0;
        m_csv << 0;
        m_csv << 0;
        m_csv << 0;
#endif
        m_csv << std::endl;
        return;
    }

    m_csv << t;
    m_csv << m_veh_state._x;
    m_csv << m_veh_state._y;
    m_csv << m_veh_state._u;
    m_csv << m_veh_state._v;
    m_csv << m_veh_state._udot;
    m_csv << m_veh_state._vdot;
    m_csv << m_veh_state._phi;
    m_csv << m_veh_state._psi;
    m_csv << m_veh_state._wx;
    m_csv << m_veh_state._wz;
    if (m_tire_type == TireType::TMeasy) {
        m_csv << m_tireTMlf_state._omega;
        m_csv << m_tireTMrf_state._omega;
        m_csv << m_tireTMlr_state._omega;
        m_csv << m_tireTMrr_state._omega;
    } else {
        m_csv << m_tireTMNrlf_state._omega;
        m_csv << m_tireTMNrrf_state._omega;
        m_csv << m_tireTMNrlr_state._omega;
        m_csv << m_tireTMNrrr_state._omega;
    }
    m_csv << m_veh_state._tor / 4.;
    m_csv << m_veh_state._current_gr + 1;
    m_csv << m_veh_state._crankOmega;
#ifdef DEBUG
    m_csv << M_DEBUG_LF_TIRE_FX;
    m_csv << M_DEBUG_RF_TIRE_FX;
    m_csv << M_DEBUG_LR_TIRE_FX;
    m_csv << M_DEBUG_RR_TIRE_FX;
    m_csv << M_DEBUG_LF_TIRE_FY;
    m_csv << M_DEBUG_RF_TIRE_FY;
    m_csv << M_DEBUG_LR_TIRE_FY;
    m_csv << M_DEBUG_RR_TIRE_FY;
    m_csv << M_DEBUG_LF_TIRE_FZ;
    m_csv << M_DEBUG_RF_TIRE_FZ;
    m_csv << M_DEBUG_LR_TIRE_FZ;
    m_csv << M_DEBUG_RR_TIRE_FZ;
#endif
    m_csv << std::endl;
}

// ======================================================================================================================

void d18SolverHalfImplicitGPU::WriteToFile() {
    if (!m_output) {
        std::cout << "No output file specified. Call SetOutput() before calling WriteToFile()" << std::endl;
        return;
    }
    m_csv.write_to_file(m_output_file);
    m_csv.clearData();
    m_timeStepsStored = 0;
}

// ======================================================================================================================

// Utility functions for finite differencing

void packY(const d18::VehicleState& v_states,
           const d18::TMeasyState& tirelf_st,
           const d18::TMeasyState& tirerf_st,
           const d18::TMeasyState& tirelr_st,
           const d18::TMeasyState& tirerr_st,
           bool has_TC,
           std::vector<double>& y) {
    int index = 0;

    // Tire deflections (lf, rf, lr and rr)

    y[index++] = tirelf_st._xe;
    y[index++] = tirelf_st._ye;
    y[index++] = tirerf_st._xe;
    y[index++] = tirerf_st._ye;
    y[index++] = tirelr_st._xe;
    y[index++] = tirelr_st._ye;
    y[index++] = tirerr_st._xe;
    y[index++] = tirerr_st._ye;

    // Wheel angular velocities (lf, rf, lr and rr)
    y[index++] = tirelf_st._omega;
    y[index++] = tirerf_st._omega;
    y[index++] = tirelr_st._omega;
    y[index++] = tirerr_st._omega;

    // Crank angular velocity - This is a state only when a torque converter is
    // used
    if (has_TC) {
        y[index++] = v_states._crankOmega;
    }

    // Vehicle states
    y[index++] = v_states._x;    // X position
    y[index++] = v_states._y;    // Y position
    y[index++] = v_states._u;    // longitudinal velocity
    y[index++] = v_states._v;    // lateral velocity
    y[index++] = v_states._psi;  // yaw angle
    y[index++] = v_states._wz;   // yaw rate
    y[index++] = v_states._phi;  // roll angle
    y[index++] = v_states._wx;   // roll rate
}

void packY(const d18::VehicleState& v_states,
           const d18::TMeasyNrState& tirelf_st,
           const d18::TMeasyNrState& tirerf_st,
           const d18::TMeasyNrState& tirelr_st,
           const d18::TMeasyNrState& tirerr_st,
           bool has_TC,
           std::vector<double>& y) {
    int index = 0;

    // Wheel angular velocities (lf, rf, lr and rr)
    y[index++] = tirelf_st._omega;
    y[index++] = tirerf_st._omega;
    y[index++] = tirelr_st._omega;
    y[index++] = tirerr_st._omega;

    // Crank angular velocity - This is a state only when a torque converter is
    // used
    if (has_TC) {
        y[index++] = v_states._crankOmega;
    }

    // Vehicle states
    y[index++] = v_states._x;    // X position
    y[index++] = v_states._y;    // Y position
    y[index++] = v_states._u;    // longitudinal velocity
    y[index++] = v_states._v;    // lateral velocity
    y[index++] = v_states._psi;  // yaw angle
    y[index++] = v_states._wz;   // yaw rate
    y[index++] = v_states._phi;  // roll angle
    y[index++] = v_states._wx;   // roll rate
}

void packYDOT(const d18::VehicleState& v_states,
              const d18::TMeasyState& tirelf_st,
              const d18::TMeasyState& tirerf_st,
              const d18::TMeasyState& tirelr_st,
              const d18::TMeasyState& tirerr_st,
              bool has_TC,
              std::vector<double>& ydot) {
    int index = 0;

    ydot[index++] = tirelf_st._xedot;
    ydot[index++] = tirelf_st._yedot;
    ydot[index++] = tirerf_st._xedot;
    ydot[index++] = tirerf_st._yedot;
    ydot[index++] = tirelr_st._xedot;
    ydot[index++] = tirelr_st._yedot;
    ydot[index++] = tirerr_st._xedot;
    ydot[index++] = tirerr_st._yedot;

    ydot[index++] = tirelf_st._dOmega;
    ydot[index++] = tirerf_st._dOmega;
    ydot[index++] = tirelr_st._dOmega;
    ydot[index++] = tirerr_st._dOmega;

    if (has_TC) {
        ydot[index++] = v_states._dOmega_crank;
    }

    ydot[index++] = v_states._dx;
    ydot[index++] = v_states._dy;
    ydot[index++] = v_states._udot;
    ydot[index++] = v_states._vdot;
    ydot[index++] = v_states._wz;
    ydot[index++] = v_states._wzdot;
    ydot[index++] = v_states._wx;
    ydot[index++] = v_states._wxdot;
}

void packYDOT(const d18::VehicleState& v_states,
              const d18::TMeasyNrState& tirelf_st,
              const d18::TMeasyNrState& tirerf_st,
              const d18::TMeasyNrState& tirelr_st,
              const d18::TMeasyNrState& tirerr_st,
              bool has_TC,
              std::vector<double>& ydot) {
    int index = 0;

    ydot[index++] = tirelf_st._dOmega;
    ydot[index++] = tirerf_st._dOmega;
    ydot[index++] = tirelr_st._dOmega;
    ydot[index++] = tirerr_st._dOmega;

    if (has_TC) {
        ydot[index++] = v_states._dOmega_crank;
    }

    ydot[index++] = v_states._dx;
    ydot[index++] = v_states._dy;
    ydot[index++] = v_states._udot;
    ydot[index++] = v_states._vdot;
    ydot[index++] = v_states._wz;
    ydot[index++] = v_states._wzdot;
    ydot[index++] = v_states._wx;
    ydot[index++] = v_states._wxdot;
}

void unpackY(const std::vector<double>& y,
             bool has_TC,
             d18::VehicleState& v_states,
             d18::TMeasyState& tirelf_st,
             d18::TMeasyState& tirerf_st,
             d18::TMeasyState& tirelr_st,
             d18::TMeasyState& tirerr_st) {
    int index = 0;
    // Tire deflections
    tirelf_st._xe = y[index++];
    tirelf_st._ye = y[index++];
    tirerf_st._xe = y[index++];
    tirerf_st._ye = y[index++];
    tirelr_st._xe = y[index++];
    tirelr_st._ye = y[index++];
    tirerr_st._xe = y[index++];
    tirerr_st._ye = y[index++];

    // Wheel angular velocities
    tirelf_st._omega = y[index++];
    tirerf_st._omega = y[index++];
    tirelr_st._omega = y[index++];
    tirerr_st._omega = y[index++];

    // Crank angular velocity - This is a state only when a torque converter is
    // used
    if (has_TC) {
        v_states._crankOmega = y[index++];
    }

    // Vehicle states
    v_states._x = y[index++];    // X position
    v_states._y = y[index++];    // Y position
    v_states._u = y[index++];    // longitudinal velocity
    v_states._v = y[index++];    // lateral velocity
    v_states._psi = y[index++];  // yaw angle
    v_states._wz = y[index++];   // yaw rate
    v_states._phi = y[index++];  // roll angle
    v_states._wx = y[index++];   // roll rate
}

void unpackY(const std::vector<double>& y,
             bool has_TC,
             d18::VehicleState& v_states,
             d18::TMeasyNrState& tirelf_st,
             d18::TMeasyNrState& tirerf_st,
             d18::TMeasyNrState& tirelr_st,
             d18::TMeasyNrState& tirerr_st) {
    int index = 0;

    // Wheel angular velocities
    tirelf_st._omega = y[index++];
    tirerf_st._omega = y[index++];
    tirelr_st._omega = y[index++];
    tirerr_st._omega = y[index++];

    // Crank angular velocity - This is a state only when a torque converter is
    // used
    if (has_TC) {
        v_states._crankOmega = y[index++];
    }

    // Vehicle states
    v_states._x = y[index++];    // X position
    v_states._y = y[index++];    // Y position
    v_states._u = y[index++];    // longitudinal velocity
    v_states._v = y[index++];    // lateral velocity
    v_states._psi = y[index++];  // yaw angle
    v_states._wz = y[index++];   // yaw rate
    v_states._phi = y[index++];  // roll angle
    v_states._wx = y[index++];   // roll rate
}