#include <hip/hip_runtime.h>
#include <iostream>
#include <random>
#include <hip/hip_runtime.h>
#include <math.h>
#include <numeric>
#include <algorithm>
#include <iterator>
#include <gtest/gtest.h>

#include "dof24_halfImplicit_gpu.cuh"

TEST(dof24_gpu, acc_test) {
    // Get total number of vehicles from command line
    unsigned int num_vehicles = 50;
    // Set the threads per block from command line
    unsigned int threads_per_block = 32;
    // Driver inputs
    std::string driver_file = "../../24dof-gpu/data/input/acc3.txt";

    // Vehicle specification
    std::string vehParamsJSON = (char*)"../../24dof-gpu/data/json/HMMWV/vehicle.json";
    std::string tireParamsJSON = (char*)"../../24dof-gpu/data/json/HMMWV/tmeasy.json";
    std::string susParamsJSON = (char*)"../../24dof-gpu/data/json/HMMWV/suspension.json";
    // Construct the solver
    d24SolverHalfImplicitGPU solver(num_vehicles);
    // The number of vehicles here sets these parameters and inputs for all these vehicles
    // If there is a need to set different parameters for different vehicles, then the solver
    // needs to be constructed for each vehicle separately (using the same sovler object)
    solver.Construct(vehParamsJSON, tireParamsJSON, susParamsJSON, num_vehicles, driver_file);

    // Set the threads per block
    solver.SetThreadsPerBlock(threads_per_block);

    // Set the time step of the solver
    solver.SetTimeStep(1e-3);

    // Now we initialize the states -> These are all set to 0 (struct initializer)
    d24GPU::VehicleState veh_st;
    d24GPU::TMeasyState tirelf_st;
    d24GPU::TMeasyState tirerf_st;
    d24GPU::TMeasyState tirelr_st;
    d24GPU::TMeasyState tirerr_st;
    d24GPU::SuspensionState suslf_st;
    d24GPU::SuspensionState susrf_st;
    d24GPU::SuspensionState suslr_st;
    d24GPU::SuspensionState susrr_st;
    // Again we initialize the same states for all vehicles
    solver.Initialize(veh_st, tirelf_st, tirerf_st, tirelr_st, tirerr_st, suslf_st, susrf_st, suslr_st, susrr_st,
                      num_vehicles);

    // Set the simulation end time -> This is a input that *must* be set by the user
    solver.SetEndTime(10.0);

    // Solve
    solver.Solve();
    double expected_final_position = 89.1625;
    d24GPU::SimState sim_state_1 = solver.GetSimState(0);
    double obtained_final_position = sim_state_1._v_states._x;
    EXPECT_NEAR(expected_final_position, obtained_final_position, 1);
}

TEST(dof24_gpu, variable_controls) {
    unsigned int num_vehicles = 1000;
    unsigned int threads_per_block = 32;

    std::string file_name_1 = "acc3";
    std::string driver_file_1 = "../../24dof-gpu/data/input/" + file_name_1 + ".txt";

    std::string file_name_2 = "double_lane4";
    std::string driver_file_2 = "../../24dof-gpu/data/input/" + file_name_2 + ".txt";

    std::string vehParamsJSON = (char*)"../../24dof-gpu/data/json/HMMWV/vehicle.json";
    std::string tireParamsJSON = (char*)"../../24dof-gpu/data/json/HMMWV/tmeasy.json";
    std::string susParamsJSON = (char*)"../../24dof-gpu/data/json/HMMWV/suspension.json";

    d24SolverHalfImplicitGPU solver(num_vehicles);

    solver.Construct(vehParamsJSON, tireParamsJSON, susParamsJSON, 500, driver_file_1);
    solver.Construct(vehParamsJSON, tireParamsJSON, susParamsJSON, 500, driver_file_2);

    solver.SetThreadsPerBlock(threads_per_block);

    solver.SetTimeStep(1e-3);

    d24GPU::VehicleState veh_st;
    d24GPU::TMeasyState tirelf_st;
    d24GPU::TMeasyState tirerf_st;
    d24GPU::TMeasyState tirelr_st;
    d24GPU::TMeasyState tirerr_st;
    d24GPU::SuspensionState suslf_st;
    d24GPU::SuspensionState susrf_st;
    d24GPU::SuspensionState suslr_st;
    d24GPU::SuspensionState susrr_st;
    // Again we initialize the same states for all vehicles
    solver.Initialize(veh_st, tirelf_st, tirerf_st, tirelr_st, tirerr_st, suslf_st, susrf_st, suslr_st, susrr_st,
                      num_vehicles);
    solver.SetEndTime(22.0);

    solver.Solve();

    d24GPU::SimState sim_state_1 = solver.GetSimState(499);
    d24GPU::SimState sim_state_2 = solver.GetSimState(999);
    double should_be1 = 235.685;
    double should_be2 = 91.5211;
    double check1 = sim_state_1._v_states._x;
    double check2 = sim_state_2._v_states._x;
    EXPECT_NEAR(check1, should_be1, 1);
    EXPECT_NEAR(check2, should_be2, 1);
}
